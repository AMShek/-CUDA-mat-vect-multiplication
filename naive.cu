
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>

//һ�κ˺������ü���һ�����C�е�Ԫ��
void __global__ MVMulCUDA(float *A,  float *B, float *C, int rowSize, int columnSize, int wA){
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	uint t_id = blockDim.x * bx+ tx;

    if(rowSize <= t_id)
		return;
    
	float Csub = 0;
    for(int i = 0; i < columnSize; i++){
		Csub += A[t_id * wA + i] * B[i];
    }
    C[t_id] = Csub;
}

void ConstantInit_A(float *data, int w, int h) {
	//row
	for (int i = 0; i < h; i++) {
		//column
		for (int j = 0; j<w; j++) {
			data[i*w + j] = i - 0.1*j + 1;
		}
	}
}

void ConstantInit_B(float *data, int h) {
	//row
	for (int i = 0; i < h; i++) {
		data[i] = log(sqrt(i*i - i + 1));
	}
}

void MVMul(float* A, float* B, float* C, int row, int column){
	int wA = 10;
	/*
	int width = column * sizeof(float);
	int	height = row;
	int columnSize = column * sizeof(float);
	int rowSize = row * sizeof(float);
	*/
	int size_A = row * column;
	int mem_size_A = sizeof(float)*size_A;
	float *h_A = (float*)malloc(mem_size_A);

	int size_B = row * 1;
	int mem_size_B = sizeof(float) * size_B;
	float *h_B = (float*)malloc(mem_size_B);

	// Allocate device memory
    float* d_A, *d_B, *d_C;
    
	// Allocate host vector C
	dim3 dimsC(row, 1, 1);
	int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
	float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));
	if (h_C == NULL) {
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	//Allocate device memory
	hipMalloc(&d_A, mem_size_A);
	hipMalloc(&d_B, mem_size_B);
	hipMalloc(&d_C, mem_size_C);
	

	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    //setup the execution configuration
    int dimGrid = 10;
    int dimBlock = 1000;

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	hipEventCreate(&start);

	hipEvent_t stop;
	hipEventCreate(&stop);

	// Record the start event
	hipEventRecord(start, NULL);

    //Launch the device computeation threads
    MVMulCUDA<<<dimGrid, dimBlock>>>(d_A, d_B, d_C,row, column, wA/sizeof(float));
	
	// Record the stop event
	hipEventRecord(stop, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);

	// Compute and print the performance
	printf("GPU running time= %.12f msec\n",msecTotal);

    //Read C from device
    hipMemcpy(C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


int show(float* C, int rowSize, int columnSize, int showsize){
    int i, j;
    if(showsize < rowSize)
        rowSize = showsize;
    if(rowSize == 1)
        columnSize = 0;
    for(i=0; i<rowSize; i++){
        for(j=0; j<showsize; j++){
            printf("%f ", C[i*columnSize + j]);
        }
        printf("\n");
    }
    return 0;
}


int main(){

    //�����ڴ�
    float* A;
    float* B;
    float* C;
    int i, j, rowSize = 10000, columnSize = 10000;
    int size = rowSize * columnSize;
    A = (float*)malloc(size * sizeof(float));
    B = (float*)malloc(columnSize * sizeof(float));
    C = (float*)malloc(rowSize * sizeof(float));
	
	//Initial matrix A and vector B
	ConstantInit_A(A, columnSize, rowSize);
	ConstantInit_B(B, rowSize);

    struct timeval tvs,tve;

    //Calculate
    hipDeviceReset(); 
    gettimeofday(&tvs,NULL);  
    MVMul(A, B, C, rowSize, columnSize);
    gettimeofday(&tve,NULL);
    hipDeviceReset();    

    //Result examples
    double span = tve.tv_sec-tvs.tv_sec + (tve.tv_usec-tvs.tv_usec)/1000000.0;
    printf("Total running time: %.12f sec\n",span);
    printf("Result examples:\n");
    show(C, 1, rowSize, 10);

    free(A);
    free(B);
    free(C);
    A = NULL;
    B = NULL;
    C = NULL;

    return 0;
}
